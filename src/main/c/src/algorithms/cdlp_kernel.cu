#include "hip/hip_runtime.h"
#include "cdlp_kernel.cuh"
#include <iostream>

//  choose to open on
#define optimized1 0                // use local bin count
#define optimized_hash 0            // use hash table for counting
#define optimized_hash1 0           // use dynamic kernel launch together with hash table
#define optimized_hash_shared 1     // use shared memory for hash table

// always open
#define optimized_skip_checkequal 1 // skip the first few check equal, since the labels won't be equal

constexpr int GRID_DIM = 64;
constexpr int BLOCK_DIM = 1024;
constexpr int LOCAL_BIN_SIZE = 16;
constexpr int PARALLEL_KERNEL_THRESHOLD = 1200;
constexpr int STOP_PARALLEL_KERNEL_ITERATIONS = 5;
constexpr int START_PARALLEL_KERNEL_ITERATIONS = 3;
constexpr int CHILD_BLOCK_DIM = 64;
constexpr int MAX_CHILD_BLOCK_DIM = 512;

constexpr int MAX_HASH_ITEMS_IN_SHARED = 1024;
constexpr int MIN_HASH_ITEMS_IN_SHARED = 128;

#define MIN(x, y) (x < y ? x : y)

__host__ __device__ static inline int ceil_div(int x, int y)
{
    return (x - 1) / y + 1;
}

// initialize labels for CDLP
__global__ void initialize_label(GrB_Index *labels, GrB_Index N)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < ceil_div(N, gridDim.x * blockDim.x); ++i)
    {
        int idx = i * gridDim.x * blockDim.x + x;
        if (idx < N)
        {
            labels[idx] = idx;
        }
    }
}

// check if two arrays are equal
// possible optimization: avoid atomicAdd
// think of this problem as a reduction problem, then optimize the memeory access
// TODO: use reduction optimizations from ECE408
__global__ void check_equality(GrB_Index *labels, GrB_Index *new_labels, GrB_Index N, int *is_equal_k)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < ceil_div(N, gridDim.x * blockDim.x); ++i)
    {
        int idx = i * gridDim.x * blockDim.x + x;
        if (idx < N && *is_equal_k)
        {
            if (labels[idx] != new_labels[idx])
            {
                atomicAnd(is_equal_k, 0);
            }
        }
    }
    __syncthreads();
}

__global__ void cdlp_base(
    GrB_Index *Ap,         // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    GrB_Index *bin_count, GrB_Index *bin_label)
{
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            for (GrB_Index j = j_base; j < j_max; j++)
            {
                GrB_Index desNode = Aj[j];
                GrB_Index label = labels[desNode]; // Label of destination node

                // 1.1 If is a directed graph
                GrB_Index incr = 1;
                // if (!symmetric) {
                //     // Find whether the arc is dual
                //     for (GrB_Index i = Ap[desNode]; i < Ap[desNode+1]; i++) {
                //         if (Aj[i] == srcNode) {
                //             incr = 2;
                //             break;
                //         }
                //     }
                // }

                // 1.2 Initalize bin & count label
                bool isNew = true;
                // Whether the label is presented in bin
                for (GrB_Index b = j_base; b < j; b++)
                {
                    if (bin_label[b] == label)
                    {
                        bin_count[b] += incr;
                        isNew = false;
                        break;
                    }
                }
                if (isNew)
                {
                    bin_label[j] = label;
                    bin_count[j] = incr;
                }
                else
                {
                    bin_label[j] = (GrB_Index)-1;
                    bin_count[j] = (GrB_Index)0;
                }
            }

            // 2. Find label with maximum frequence
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;
            for (GrB_Index j = j_base; j < j_max; j++)
            {
                if (max_count < bin_count[j])
                {
                    max_count = bin_count[j];
                    min_label = bin_label[j];
                }
                else if (max_count == bin_count[j] && min_label > bin_label[j] && bin_label[j] != (GrB_Index)-1)
                {
                    min_label = bin_label[j];
                }
                else
                {
                }
            }

            // 3. Update label
            if (min_label != (GrB_Index)-1)
            {
                // labels[srcNode] = min_label; // TODO: potential overflow
                new_labels[srcNode] = min_label;
            }
        }
        // __syncthreads();
    }
}

__global__ void cdlp_optimized1(
    GrB_Index *Ap,         // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    GrB_Index *bin_count, GrB_Index *bin_label, int *bin_index)
{
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            GrB_Index local_bin_count[LOCAL_BIN_SIZE];
            GrB_Index local_bin_label[LOCAL_BIN_SIZE];
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            auto neighbor_n = j_max - j_base;
            auto local_n = min(static_cast<unsigned long>(LOCAL_BIN_SIZE), static_cast<unsigned long>(neighbor_n));
            GrB_Index bin_base = j_base;
            for (GrB_Index j = 0; j < local_n; j++)
            {
                GrB_Index desNode = Aj[j + j_base];
                GrB_Index label = labels[desNode]; // Label of destination node

                // 1.1 If is a directed graph
                GrB_Index incr = 1;

                // 1.2 Initalize bin & count label
                bool isNew = true;
                // Whether the label is presented in bin
                for (GrB_Index b = 0; b < j; b++)
                {
                    if (local_bin_label[b] == label)
                    {
                        local_bin_count[b] += incr;
                        isNew = false;
                        break;
                    }
                }
                if (isNew)
                {
                    local_bin_label[j] = label;
                    local_bin_count[j] = incr;
                }
                else
                {
                    local_bin_label[j] = (GrB_Index)-1;
                    local_bin_count[j] = (GrB_Index)0;
                }
            }
            if (neighbor_n > LOCAL_BIN_SIZE)
            {
                // For next optimization, parallelize this part if neighbor_n >> LOCAL_BIN_SIZE
                // bin_base = atomicAdd(bin_index, neighbor_n - LOCAL_BIN_SIZE); // allocate space in the global bin
                for (GrB_Index j = 0; j < neighbor_n - LOCAL_BIN_SIZE; j++)
                {
                    GrB_Index desNode = Aj[j + LOCAL_BIN_SIZE + j_base];
                    GrB_Index label = labels[desNode]; // Label of destination node

                    // 1.1 If is a directed graph
                    GrB_Index incr = 1;

                    // 1.2 Initalize bin & count label
                    bool isNew = true;
                    // Whether the label is presented in local bin
                    for (GrB_Index b = 0; b < LOCAL_BIN_SIZE; b++)
                    {
                        if (local_bin_label[b] == label)
                        {
                            local_bin_count[b] += incr;
                            isNew = false;
                            break;
                        }
                    }
                    if (isNew)
                    {
                        for (GrB_Index b = bin_base; b < bin_base + j; b++)
                        {
                            if (bin_label[b] == label)
                            {
                                bin_count[b] += incr;
                                isNew = false;
                                break;
                            }
                        }
                        auto b = bin_base + j;
                        if (isNew)
                        {
                            bin_label[b] = label;
                            bin_count[b] = incr;
                        }
                        else
                        {
                            bin_label[b] = (GrB_Index)-1;
                            bin_count[b] = (GrB_Index)0;
                        }
                    }
                    else
                    {
                        auto b = bin_base + j;
                        bin_label[b] = (GrB_Index)-1;
                        bin_count[b] = (GrB_Index)0;
                    }
                }
            }

            // 2. Find label with maximum frequence
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;
            for (GrB_Index j = 0; j < local_n; j++)
            {
                if (max_count < local_bin_count[j])
                {
                    max_count = local_bin_count[j];
                    min_label = local_bin_label[j];
                }
                else if (max_count == local_bin_count[j] && min_label > local_bin_label[j] && local_bin_label[j] != (GrB_Index)-1)
                {
                    min_label = local_bin_label[j];
                }
            }
            if (neighbor_n > LOCAL_BIN_SIZE)
            {
                for (GrB_Index j = bin_base; j < bin_base + neighbor_n - LOCAL_BIN_SIZE; j++)
                {
                    if (max_count < bin_count[j])
                    {
                        max_count = bin_count[j];
                        min_label = bin_label[j];
                    }
                    else if (max_count == bin_count[j] && min_label > bin_label[j] && bin_label[j] != (GrB_Index)-1)
                    {
                        min_label = bin_label[j];
                    }
                }
            }

            // 3. Update label
            new_labels[srcNode] = min_label;
        }
    }
}

// hash table optimization
#define HASH_TABLE_SIZE_FACTOR 2

typedef struct
{
#if optimized_hash1 || optimized_hash_shared
    int mutex;
#endif
    int iter_count;   // version number
    // int relative_idx; // index number in the bin
    GrB_Index label;  // key for hashing
    GrB_Index count;  // value for hashing
} hash_table_item;

// hash function for key = label and capacity = n
__device__ __forceinline__ int hash_func(GrB_Index label, GrB_Index n)
{
    const GrB_Index prime = 2147483647; // A large prime number
    GrB_Index hash = label % prime;
    hash %= n;
    return (int)hash;
}

__device__ __forceinline__ GrB_Index hash_table_get_count(int iter, hash_table_item *htable, GrB_Index start, GrB_Index end, GrB_Index label)
{
    return 0;
}

// __device__ __forceinline__ GrB_Index hash_table_set_count(int iter, hash_table_item* htable, GrB_Index start, GrB_Index end, GrB_Index label, GrB_Index value)
// {
//     return 0;
// }

__device__ __forceinline__ GrB_Index hash_table_inc_count(int iter, hash_table_item *htable, GrB_Index start, GrB_Index end, GrB_Index label, GrB_Index delta)
{
#define gethtable(idx) htable[start + idx]
    GrB_Index capacity = end - start + 1; // capacity should not overflow int32_t even though we use uint64_t, otherwise too large
    int relative_idx = hash_func(label, capacity);
    int location = -1;
    // linear probing
    while (gethtable(relative_idx).iter_count == iter)
    {
        // same iteration number, key already exist, add count
        if (gethtable(relative_idx).label == label)
        {
            gethtable(relative_idx).count += delta;
            return gethtable(relative_idx).count;
        }
        relative_idx = (relative_idx + 1) % capacity;
    }
    // break out of the loop, key not exist, add new key
    if (location < 0)
    {
        location = relative_idx;
    }
    gethtable(location).iter_count = iter;
    // gethtable(location).relative_idx = relative_idx;
    gethtable(location).label = label;
    gethtable(location).count = delta;
    return gethtable(location).count;
#undef gethtable
}

#if optimized_hash1
__device__ __forceinline__ GrB_Index hash_table_inc_count_atomic(int iter, hash_table_item *htable, GrB_Index start, GrB_Index end, GrB_Index label, GrB_Index delta)
{
#define gethtable(idx) htable[start + idx]
#define lockslot(idx) while (atomicCAS(&(gethtable(idx).mutex), 0, 1) != 0)
#define unlockslot(idx) atomicExch(&(gethtable(idx).mutex), 0);
    GrB_Index capacity = end - start + 1; // capacity should not overflow int32_t even though we use uint64_t, otherwise too large
    int relative_idx = hash_func(label, capacity);
    int location = -1;
    // linear probing
    while (1)
    {
        lockslot(relative_idx);
        if (gethtable(relative_idx).iter_count != iter){
            break;
        }
        // same iteration number, key already exist, add count
        if (gethtable(relative_idx).label == label)
        {
            // gethtable(relative_idx).count += delta;
            gethtable(relative_idx).count += delta;
            auto ret = gethtable(relative_idx).count;
            unlockslot(relative_idx);
            // return gethtable(relative_idx).count;
            return ret;
        }
        unlockslot(relative_idx);
        relative_idx = (relative_idx + 1) % capacity;
    }
    // break out of the loop, key not exist, add new key
    if (location < 0)
    {
        location = relative_idx;
    }
    gethtable(location).iter_count = iter;
    // gethtable(location).relative_idx = location;
    gethtable(location).label = label;
    gethtable(location).count = delta;
    unlockslot(location);
    return delta;
#undef unlockslot
#undef lockslot
#undef gethtable
}
#endif

__global__ void cdlp_base_with_hashing(
    int iteration_count,   // Current iteration, start counting from 1 so that 0 means not yet used
    GrB_Index *Ap,         // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    hash_table_item *htable)
{
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;

            for (GrB_Index j = j_base; j < j_max; j++)
            {
                GrB_Index desNode = Aj[j];
                GrB_Index label = labels[desNode]; // Label of destination node

                // 1.1 If is a directed graph
                GrB_Index incr = 1;

                // 1.2 build hash table
                int segment_start = j_base * HASH_TABLE_SIZE_FACTOR;
                int segment_end = j_max * HASH_TABLE_SIZE_FACTOR - 1; // inclusive index
                GrB_Index new_count = hash_table_inc_count(iteration_count, htable, segment_start, segment_end, label, incr);
                if (new_count > max_count)
                {
                    max_count = new_count;
                    min_label = label;
                }
                else if (new_count == max_count && label < min_label)
                {
                    min_label = label;
                }
            }

            // 2. Update label
            if (min_label != (GrB_Index)-1)
            {
                // labels[srcNode] = min_label; // TODO: potential overflow
                new_labels[srcNode] = min_label;
            }
        }
        // __syncthreads();
    }
}

#if optimized_hash1
__global__ void cdlp_child(GrB_Index srcNode, GrB_Index neighbor_n, int iteration_count, GrB_Index *Aj, GrB_Index *labels, GrB_Index *new_labels, GrB_Index j_base, GrB_Index j_max, hash_table_item *htable)
{
    int ti = threadIdx.x;
    unsigned long long max_count = (unsigned long long)0;
    unsigned long long min_label = (unsigned long long)-1;
    int segment_start = j_base * HASH_TABLE_SIZE_FACTOR;
    int segment_end = j_max * HASH_TABLE_SIZE_FACTOR - 1; // inclusive index
    // use int instead of GrB_Index to save resource, the count won't really exceed int anyway
    __shared__ unsigned long long shared_min_label;
    __shared__ unsigned long long shared_max_count;
    if (ti == 0){
        shared_min_label = (unsigned long long)-1;
        shared_max_count = (unsigned long long)0;
    }
    __syncthreads();
    // start to update max count
    for (GrB_Index j = ti; j < neighbor_n; j += blockDim.x)
    {
        GrB_Index desNode = Aj[j_base + j];
        GrB_Index label = labels[desNode]; // Label of destination node

        // 1.1 If is a directed graph
        GrB_Index incr = 1;

        // 1.2 build hash table
        GrB_Index new_count = hash_table_inc_count_atomic(iteration_count, htable, segment_start, segment_end, label, incr);

        if (new_count > max_count)
        {
            max_count = new_count;
            min_label = label;
        }
        else if (new_count == max_count && label < min_label)
        {
            min_label = label;
        }
    }
    atomicMax(&shared_max_count, max_count);
    __syncthreads();
    if (max_count == shared_max_count)
    {
        atomicMin(&shared_min_label, min_label);
    }
    __syncthreads();
    if (ti == 0){
        new_labels[srcNode] = shared_min_label;
    }
}


__global__ void cdlp_base_with_hashing1(
    int iteration_count,   // Current iteration, start counting from 1 so that 0 means not yet used
    GrB_Index *Ap,         // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    hash_table_item *htable
){
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;
            GrB_Index neighbor_n = j_max - j_base;
            if (neighbor_n > PARALLEL_KERNEL_THRESHOLD && iteration_count < STOP_PARALLEL_KERNEL_ITERATIONS)
            {
                new_labels[srcNode] = (GrB_Index)-1;
                int blocksize_dynamic = ceil_div(neighbor_n, PARALLEL_KERNEL_THRESHOLD) * CHILD_BLOCK_DIM;
                cdlp_child<<<1, blocksize_dynamic>>>(srcNode, neighbor_n, iteration_count, Aj, labels, new_labels, j_base, j_max, htable);
                hipDeviceSynchronize();
            }
            else
            {
                // 1.2 build hash table
                int segment_start = j_base * HASH_TABLE_SIZE_FACTOR;
                int segment_end = j_max * HASH_TABLE_SIZE_FACTOR - 1; // inclusive index
                for (GrB_Index j = j_base; j < j_max; j++)
                {
                    GrB_Index desNode = Aj[j];
                    GrB_Index label = labels[desNode]; // Label of destination node

                    // 1.1 If is a directed graph
                    GrB_Index incr = 1;

                    GrB_Index new_count = hash_table_inc_count(iteration_count, htable, segment_start, segment_end, label, incr);
                    if (new_count > max_count)
                    {
                        max_count = new_count;
                        min_label = label;
                    }
                    else if (new_count == max_count && label < min_label)
                    {
                        min_label = label;
                    }
                }

                // 2. Update label
                if (min_label != (GrB_Index)-1)
                {
                    // labels[srcNode] = min_label; // TODO: potential overflow
                    new_labels[srcNode] = min_label;
                }
            }
        }
        // __syncthreads();
    }
}
#endif

#if optimized_hash_shared

__device__ __forceinline__ float get_shared_mem_utilization(int size, int used){
    return (float)used / size;
}

// return -1 on failure (maybe more that 75% full)
__device__ __forceinline__ int inc_count_atomic_shared_htable(int iter, hash_table_item* shared_htable, int sharedmem_slots, int* shared_mem_usage_ptr, GrB_Index label, GrB_Index delta){
#define getsharedhtable(idx) shared_htable[idx]
#define lockslot(idx) while (atomicCAS(&(getsharedhtable(idx).mutex), 0, 1) != 0)
#define unlockslot(idx) atomicExch(&(getsharedhtable(idx).mutex), 0);
    int relative_idx = hash_func(label, (GrB_Index)sharedmem_slots);
    int location = -1;
    // linear probing
    while (1)
    {
        lockslot(relative_idx);
        if (getsharedhtable(relative_idx).iter_count != iter){
            // find a free slot, check capacity then
            if (get_shared_mem_utilization(atomicAdd(shared_mem_usage_ptr, 0), sharedmem_slots) > 0.75){
                unlockslot(relative_idx);
                return -1;
            }
            break;
        }
        // same iteration number, key already exist, add count
        if (getsharedhtable(relative_idx).label == label)
        {
            // gethtable(relative_idx).count += delta;
            getsharedhtable(relative_idx).count += delta;
            auto ret = getsharedhtable(relative_idx).count;
            unlockslot(relative_idx);
            // return gethtable(relative_idx).count;
            return ret;
        }
        unlockslot(relative_idx);
        relative_idx = (relative_idx + 1) % sharedmem_slots;
    }
    // break out of the loop, key not exist, add new key
    if (location < 0)
    {
        location = relative_idx;
    }
    atomicAdd(shared_mem_usage_ptr, 1);
    getsharedhtable(location).iter_count = iter;
    // gethtable(location).relative_idx = location;
    getsharedhtable(location).label = label;
    getsharedhtable(location).count = delta;
    unlockslot(location);
    return delta;
#undef unlockslot
#undef lockslot
#undef getsharedhtable
}

// two level hash table design: shared mem + global mem
// intuition is that, as label converges, most of the label will be the same, so we can use shared mem to store the most frequent labels
// for every incoming inc_count operation, first check if it in the shared mem
// if it's in the shared mem, then increment it
// if it's not, then check the capacity usage of shared mem, if it's not full, then insert it into shared mem
// otherwise insert it to global mem
// tradeoff: for every inc_count, whose label is in global mem, we have to check if it's inside shared mem first, then check if it's in global mem
__device__ __forceinline__ GrB_Index hash_table_inc_count_atomic_sharedmem(int iter, hash_table_item *htable, hash_table_item *shared_htable, int sharedmem_size, int* shared_mem_usage_ptr, GrB_Index start, GrB_Index end, GrB_Index label, GrB_Index delta)
{
#define gethtable(idx) htable[start + idx]
#define lockslot(idx) while (atomicCAS(&(gethtable(idx).mutex), 0, 1) != 0)
#define unlockslot(idx) atomicExch(&(gethtable(idx).mutex), 0);
    GrB_Index capacity = end - start + 1; // capacity should not overflow int32_t even though we use uint64_t, otherwise too large
    // try shared mem first
    // int ret = -1;
    // ret = inc_count_atomic_shared_htable(iter, shared_htable, sharedmem_size/sizeof(hash_table_item), shared_mem_usage_ptr, label, delta);
    // if (ret >= 0){
    //     return ret;
    // }
    int relative_idx = hash_func(label, capacity);
    int location = -1;
    // linear probing
    while (1)
    {
        lockslot(relative_idx);
        if (gethtable(relative_idx).iter_count != iter){
            break;
        }
        // same iteration number, key already exist, add count
        if (gethtable(relative_idx).label == label)
        {
            // gethtable(relative_idx).count += delta;
            gethtable(relative_idx).count += delta;
            auto ret = gethtable(relative_idx).count;
            unlockslot(relative_idx);
            // return gethtable(relative_idx).count;
            return ret;
        }
        unlockslot(relative_idx);
        relative_idx = (relative_idx + 1) % capacity;
    }
    // break out of the loop, key not exist, add new key
    if (location < 0)
    {
        location = relative_idx;
    }
    gethtable(location).iter_count = iter;
    // gethtable(location).relative_idx = location;
    gethtable(location).label = label;
    gethtable(location).count = delta;
    unlockslot(location);
    return delta;

#undef unlockslot
#undef lockslot
#undef gethtable
}

__global__ void cdlp_child_sharedmem(GrB_Index srcNode, GrB_Index neighbor_n, int iteration_count, GrB_Index *Aj, GrB_Index *labels, GrB_Index *new_labels, GrB_Index j_base, GrB_Index j_max, hash_table_item *htable, int sharedmem_size)
{
    extern __shared__ hash_table_item shared_htable[];
    int ti = threadIdx.x;
    unsigned long long max_count = (unsigned long long)0;
    unsigned long long min_label = (unsigned long long)-1;
    int segment_start = j_base * HASH_TABLE_SIZE_FACTOR;
    int segment_end = j_max * HASH_TABLE_SIZE_FACTOR - 1; // inclusive index
    // use int instead of GrB_Index to save resource, the count won't really exceed int anyway
    __shared__ unsigned long long shared_min_label;
    __shared__ unsigned long long shared_max_count;
    __shared__ int shared_htable_usage;
    if (ti == 0){
        shared_min_label = (unsigned long long)-1;
        shared_max_count = (unsigned long long)0;
        shared_htable_usage = 0;
    }
    // clear shared mem
    int numslots = sharedmem_size / sizeof(hash_table_item);
    for (int i = ti; i < numslots; i += blockDim.x){
        shared_htable[i].iter_count = 0;
        shared_htable[i].mutex = 0;
    }
    __syncthreads();
    // start to update max count
    for (GrB_Index j = ti; j < neighbor_n; j += blockDim.x)
    {
        GrB_Index desNode = Aj[j_base + j];
        GrB_Index label = labels[desNode]; // Label of destination node

        // 1.1 If is a directed graph
        GrB_Index incr = 1;

        // 1.2 build hash table
        GrB_Index new_count = hash_table_inc_count_atomic_sharedmem(iteration_count, htable, (hash_table_item*)shared_htable, sharedmem_size, &shared_htable_usage, segment_start, segment_end, label, incr);

        if (new_count > max_count)
        {
            max_count = new_count;
            min_label = label;
        }
        else if (new_count == max_count && label < min_label)
        {
            min_label = label;
        }
    }
    atomicMax(&shared_max_count, max_count);
    __syncthreads();
    if (max_count == shared_max_count)
    {
        atomicMin(&shared_min_label, min_label);
    }
    __syncthreads();
    if (ti == 0){
        new_labels[srcNode] = shared_min_label;
    }
}


__global__ void cdlp_base_with_hashing_sharedmem(
    int iteration_count,   // Current iteration, start counting from 1 so that 0 means not yet used
    GrB_Index *Ap,         // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    hash_table_item *htable
){
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;
            GrB_Index neighbor_n = j_max - j_base;
            if (neighbor_n > PARALLEL_KERNEL_THRESHOLD && iteration_count > START_PARALLEL_KERNEL_ITERATIONS)
            {
                new_labels[srcNode] = (GrB_Index)-1;
                int blocksize_dynamic = MIN(ceil_div(neighbor_n, PARALLEL_KERNEL_THRESHOLD) * CHILD_BLOCK_DIM, MAX_CHILD_BLOCK_DIM);
                int sharedmem_size = MIN(ceil_div(neighbor_n, PARALLEL_KERNEL_THRESHOLD) * MIN_HASH_ITEMS_IN_SHARED, MAX_HASH_ITEMS_IN_SHARED) * sizeof(hash_table_item);
                cdlp_child_sharedmem<<<1, blocksize_dynamic, sharedmem_size>>>(srcNode, neighbor_n, iteration_count, Aj, labels, new_labels, j_base, j_max, htable, sharedmem_size);
                hipDeviceSynchronize();
            }
            else
            {
                // 1.2 build hash table
                int segment_start = j_base * HASH_TABLE_SIZE_FACTOR;
                int segment_end = j_max * HASH_TABLE_SIZE_FACTOR - 1; // inclusive index
                for (GrB_Index j = j_base; j < j_max; j++)
                {
                    GrB_Index desNode = Aj[j];
                    GrB_Index label = labels[desNode]; // Label of destination node

                    // 1.1 If is a directed graph
                    GrB_Index incr = 1;

                    GrB_Index new_count = hash_table_inc_count(iteration_count, htable, segment_start, segment_end, label, incr);
                    if (new_count > max_count)
                    {
                        max_count = new_count;
                        min_label = label;
                    }
                    else if (new_count == max_count && label < min_label)
                    {
                        min_label = label;
                    }
                }

                // 2. Update label
                if (min_label != (GrB_Index)-1)
                {
                    // labels[srcNode] = min_label; // TODO: potential overflow
                    new_labels[srcNode] = min_label;
                }
            }
        }
        // __syncthreads();
    }
}
#endif

// entry point
__host__ void cdlp_gpu(GrB_Index *Ap, GrB_Index Ap_size, GrB_Index *Aj, GrB_Index Aj_size, GrB_Vector *CDLP_handle, GrB_Index N, GrB_Index nnz, bool symmetric, int itermax)
{
    GrB_Index *Ap_k;
    GrB_Index *Aj_k;
    GrB_Index *labels_k;
    GrB_Index *new_labels_k;
    GrB_Index *labels;
    int is_equal = 1;
    int *is_equal_k;
#if optimized_hash || optimized_hash1 || optimized_hash_shared
    hash_table_item *htable_k;
#if DEBUG_PRINT
    hash_table_item *htable;
#endif
#else
    GrB_Index *bin_count_k, *bin_label_k; // For dynamically counting labels (can be optimized using shared memory plus a overflow global memory)
#endif

    hipMalloc((void **)&Ap_k, Ap_size);
    hipMalloc((void **)&Aj_k, Aj_size);
    hipMalloc((void **)&labels_k, N * sizeof(GrB_Index));
    hipMalloc((void **)&new_labels_k, N * sizeof(GrB_Index));
    hipHostMalloc((void **)&labels, N * sizeof(GrB_Index));
    hipMalloc((void **)&is_equal_k, sizeof(int));
#if optimized_hash || optimized_hash1 || optimized_hash_shared
    hipMalloc((void **)&htable_k, HASH_TABLE_SIZE_FACTOR * nnz * sizeof(hash_table_item));
#if DEBUG_PRINT
    hipHostMalloc((void **)&htable, HASH_TABLE_SIZE_FACTOR * nnz * sizeof(hash_table_item));
#endif
#else
    hipMalloc((void **)&bin_count_k, nnz * sizeof(GrB_Index));
    hipMalloc((void **)&bin_label_k, nnz * sizeof(GrB_Index));
#endif

// #if optimized_hash1
//     GrB_Index *max_counts_k;
//     hipMalloc((void **)&max_counts_k, N * sizeof(GrB_Index));
//     hipMemset(max_counts_k, 0, N * sizeof(GrB_Index));
// #endif

#if DEBUG_PRINT != 0
    // PRINT("FINISH CUDA MALLOC");
    std::cout << "FINISH CUDA MALLOC" << std::endl;
#endif

#if optimized1
    int *bin_index;
    hipMalloc((void **)&bin_index, sizeof(int));
    hipMemset(bin_index, 0, sizeof(int));
#endif

    hipMemcpy(Ap_k, Ap, Ap_size, hipMemcpyHostToDevice);
    hipMemcpy(Aj_k, Aj, Aj_size, hipMemcpyHostToDevice);

#if optimized_hash || optimized_hash1 || optimized_hash_shared
    hipMemset(htable_k, 0, nnz * sizeof(hash_table_item));
#endif

#if DEBUG_PRINT != 0
    // PRINT("FINISH CUDA MEMCPY");
    std::cout << "FINISH CUDA MEMCPY" << std::endl;
#endif

    dim3 DimGrid(GRID_DIM, 1, 1);
    dim3 DimBlock(BLOCK_DIM, 1);

    initialize_label<<<DimGrid, DimBlock>>>(labels_k, N);

    // timer_start("CDLP_GPU MAIN LOOP USING CUDA KERNEL");
    for (int i = 0; i < itermax; ++i)
    {
        timer_start("CDLP_GPU ITERATION " + std::to_string(i));
        // PRINT("RUNNING ITERATION {}", i);
#if DEBUG_PRINT != 0
        std::cout << "RUNNING ITERATION " << i << std::endl;
#endif

#if optimized1
        cdlp_optimized1<<<DimGrid, DimBlock>>>(Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, bin_count_k, bin_label_k, bin_index);
#elif optimized_hash
        cdlp_base_with_hashing<<<DimGrid, DimBlock>>>(i + 1, Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, htable_k);
#elif optimized_hash1
        cdlp_base_with_hashing1<<<DimGrid, DimBlock>>>(i + 1, Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, htable_k);
#elif optimized_hash_shared
        cdlp_base_with_hashing_sharedmem<<<DimGrid, DimBlock>>>(i + 1, Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, htable_k);
#else
        cdlp_base<<<DimGrid, DimBlock>>>(Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, bin_count_k, bin_label_k);
#endif

#if optimized_skip_checkequal
        if (i < MIN(itermax, 5)){
            is_equal = 0;
        }else{
            hipMemset(is_equal_k, 1, sizeof(int));
            check_equality<<<DimGrid, DimBlock>>>(labels_k, new_labels_k, N, is_equal_k);
            // hipDeviceSynchronize();
            hipMemcpy(&is_equal, is_equal_k, sizeof(int), hipMemcpyDeviceToHost);
        }
#else
        hipMemset(is_equal_k, 1, sizeof(int));
        check_equality<<<DimGrid, DimBlock>>>(labels_k, new_labels_k, N, is_equal_k);
        // hipDeviceSynchronize();
        hipMemcpy(&is_equal, is_equal_k, sizeof(int), hipMemcpyDeviceToHost);
#endif
        if (is_equal){
            break;
        }else{
            // hipMemcpy(labels_k, new_labels_k, N * sizeof(GrB_Index), hipMemcpyDeviceToDevice);
            //  optimization: double buffering, avoid memcpy
            std::swap(labels_k, new_labels_k);
#if DEBUG_PRINT
            if (N < 20)
            {
                hipMemcpy(labels, labels_k, N * sizeof(GrB_Index), hipMemcpyDeviceToHost);
                for (int j = 0; j < N; ++j)
                {
                    std::cout << labels[j] << " ";
                }
                std::cout << '\n';
                hipMemcpy(labels, max_counts_k, N * sizeof(GrB_Index), hipMemcpyDeviceToHost);
                for (int j = 0; j < N; ++j)
                {
                    std::cout << labels[j] << " ";
                }
                std::cout << '\n';
                hipMemcpy(htable, htable_k, HASH_TABLE_SIZE_FACTOR * nnz * sizeof(hash_table_item), hipMemcpyDeviceToHost);
                for (int j = 0; j < HASH_TABLE_SIZE_FACTOR * nnz; ++j)
                {
                    std::cout << htable[j].label << '@' << htable[j].count << " ";
                }
                std::cout << '\n';
            }
#endif

// #if optimized_hash1
//             // hipMemset(max_counts_k, 0, N * sizeof(GrB_Index));
//             // hipMemset(htable_k, 0, HASH_TABLE_SIZE_FACTOR * nnz * sizeof(hash_table_item));
// #endif
        }
        timer_stop();
    }
    hipDeviceSynchronize();

    // timer_stop();

    hipMemcpy(labels, labels_k, N * sizeof(GrB_Index), hipMemcpyDeviceToHost);

#if DEBUG_PRINT != 0
    // PRINT("RUNNING CUDA FREE");
    std::cout << "RUNNING CUDA FREE" << std::endl;
#endif

    hipFree(Ap_k);
    hipFree(Aj_k);
    hipFree(labels_k);
    hipFree(new_labels_k);
    hipFree(is_equal_k);
#if optimized_hash || optimized_hash1 || optimized_hash_shared
    hipFree(htable_k);
#if DEBUG_PRINT
    hipHostFree(htable);
#endif
#else
    hipFree(bin_count_k);
    hipFree(bin_label_k);
#endif
#if optimized1
    hipFree(bin_index);
#endif
// #if optimized_hash1
//     hipFree(max_counts_k);
// #endif

#if DEBUG_PRINT != 0
    // PRINT("CONVERT TO GRB_VECTOR");
    std::cout << "CONVERT TO GRB_VECTOR" << std::endl;
#endif

    GrB_Vector CDLP = NULL;
    GrB_Vector_new(&CDLP, GrB_UINT64, N);
    for (GrB_Index i = 0; i < N; i++)
    {
        GrB_Vector_setElement_UINT64(CDLP, labels[i], i);
    }
    (*CDLP_handle) = CDLP;

    hipHostFree(labels);
}