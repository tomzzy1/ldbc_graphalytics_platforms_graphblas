#include "hip/hip_runtime.h"
#include "cdlp_kernel.cuh"
#include <iostream>

// #define optimized1 1

constexpr int GRID_DIM = 256;
constexpr int BLOCK_DIM = 1024;
constexpr int LOCAL_BIN_SIZE = 1;

__host__ __device__ static inline int ceil_div(int x, int y)
{
    return (x - 1) / y + 1;
}

// initialize labels for CDLP
__global__ void initialize_label(GrB_Index *labels, GrB_Index N)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < ceil_div(N, gridDim.x * blockDim.x); ++i)
    {
        int idx = i * gridDim.x * blockDim.x + x;
        if (idx < N)
        {
            labels[idx] = idx;
        }
    }
}

// check if two arrays are equal
// possible optimization: avoid atomicAdd
// think of this problem as a reduction problem, then optimize the memeory access
// TODO: use reduction optimizations from ECE408
__global__ void check_equality(GrB_Index *labels, GrB_Index *new_labels, GrB_Index N, int *is_equal_k)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < ceil_div(N, gridDim.x * blockDim.x); ++i)
    {
        int idx = i * gridDim.x * blockDim.x + x;
        if (idx < N && *is_equal_k)
        {
            if (labels[idx] != new_labels[idx])
            {
                atomicAnd(is_equal_k, 0);
            }
        }
    }
    __syncthreads();
}

__global__ void cdlp_base(
    GrB_Index *Ap, // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    GrB_Index *bin_count, GrB_Index *bin_label)
{
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            for (GrB_Index j = j_base; j < j_max; j++)
            {
                GrB_Index desNode = Aj[j];
                GrB_Index label = labels[desNode]; // Label of destination node

                // 1.1 If is a directed graph
                GrB_Index incr = 1;
                // if (!symmetric) {
                //     // Find whether the arc is dual
                //     for (GrB_Index i = Ap[desNode]; i < Ap[desNode+1]; i++) {
                //         if (Aj[i] == srcNode) {
                //             incr = 2;
                //             break;
                //         }
                //     }
                // }

                // 1.2 Initalize bin & count label
                bool isNew = true;
                // Whether the label is presented in bin
                for (GrB_Index b = j_base; b < j; b++)
                {
                    if (bin_label[b] == label)
                    {
                        bin_count[b] += incr;
                        isNew = false;
                        break;
                    }
                }
                if (isNew)
                {
                    bin_label[j] = label;
                    bin_count[j] = incr;
                }
                else
                {
                    bin_label[j] = (GrB_Index)-1;
                    bin_count[j] = (GrB_Index)0;
                }
            }

            // 2. Find label with maximum frequence
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;
            for (GrB_Index j = j_base; j < j_max; j++)
            {
                if (max_count < bin_count[j])
                {
                    max_count = bin_count[j];
                    min_label = bin_label[j];
                }
                else if (max_count == bin_count[j] && min_label > bin_label[j] && bin_label[j] != (GrB_Index)-1)
                {
                    min_label = bin_label[j];
                }
                else
                {
                }
            }

            // 3. Update label
            if (min_label != (GrB_Index)-1)
            {
                // labels[srcNode] = min_label; // TODO: potential overflow
                new_labels[srcNode] = min_label;
            }
        }
        // __syncthreads();
    }
}

__global__ void cdlp_optimized1(
    GrB_Index *Ap, // Row pointers
    GrB_Index *Aj,         // Column indices
    GrB_Index *labels,     // Labels for each node
    GrB_Index *new_labels, // new labels after each iteration
    GrB_Index N,           // Number of nodes
    bool symmetric,        // Is the matrix symmetric (aka is the graph undirected)
    GrB_Index *bin_count, GrB_Index *bin_label, int *bin_index)
{
    GrB_Index ti = blockDim.x * blockIdx.x + threadIdx.x;
    // Iterate until converge or reaching maximum number
    // Loop through all nodes
    GrB_Index stride = gridDim.x * blockDim.x;
    for (GrB_Index srcNode = ti; srcNode < N; srcNode += stride)
    {
        if (srcNode < N)
        {
            GrB_Index local_bin_count[LOCAL_BIN_SIZE];
            GrB_Index local_bin_label[LOCAL_BIN_SIZE];
            // 1. Count neighbors' labels
            GrB_Index j_base = Ap[srcNode];
            GrB_Index j_max = Ap[srcNode + 1];
            auto neighbor_n = j_max - j_base;
            auto local_n = min(static_cast<unsigned long>(LOCAL_BIN_SIZE), static_cast<unsigned long>(neighbor_n));
            GrB_Index bin_base;
            for (GrB_Index j = 0; j < local_n; j++)
            {
                GrB_Index desNode = Aj[j + j_base];
                GrB_Index label = labels[desNode]; // Label of destination node

                // 1.1 If is a directed graph
                GrB_Index incr = 1;

                // 1.2 Initalize bin & count label
                bool isNew = true;
                // Whether the label is presented in bin
                for (GrB_Index b = 0; b < j; b++)
                {
                    if (local_bin_label[b] == label)
                    {
                        local_bin_count[b] += incr;
                        isNew = false;
                        break;
                    }
                }
                if (isNew)
                {
                    local_bin_label[j] = label;
                    local_bin_count[j] = incr;
                }
                else
                {
                    local_bin_label[j] = (GrB_Index)-1;
                    local_bin_count[j] = (GrB_Index)0;
                }
            }
            if (neighbor_n > LOCAL_BIN_SIZE)
            {
                // For next optimization, parallelize this part if neighbor_n >> LOCAL_BIN_SIZE
                bin_base = atomicAdd(bin_index, neighbor_n - LOCAL_BIN_SIZE); // allocate space in the global bin
                for (GrB_Index j = 0; j < neighbor_n - LOCAL_BIN_SIZE; j++)
                {
                    GrB_Index desNode = Aj[j + LOCAL_BIN_SIZE + j_base];
                    GrB_Index label = labels[desNode]; // Label of destination node

                    // 1.1 If is a directed graph
                    GrB_Index incr = 1;

                    // 1.2 Initalize bin & count label
                    bool isNew = true;
                    // Whether the label is presented in local bin
                    for (GrB_Index b = 0; b < LOCAL_BIN_SIZE; b++)
                    {
                        if (local_bin_label[b] == label)
                        {
                            local_bin_count[b] += incr;
                            isNew = false;
                            break;
                        }
                    }
                    if (isNew)
                    {
                        for (GrB_Index b = bin_base; b < bin_base + j; b++)
                        {
                            if (bin_label[b] == label)
                            {
                                bin_count[b] += incr;
                                isNew = false;
                                break;
                            }
                        }
                        auto b = bin_base + j;
                        if (isNew)
                        {
                            bin_label[b] = label;
                            bin_count[b] = incr;
                        }
                        else
                        {
                            bin_label[b] = (GrB_Index)-1;
                            bin_count[b] = (GrB_Index)0;
                        }
                    }
                }
            }

            // 2. Find label with maximum frequence
            GrB_Index max_count = (GrB_Index)0;
            GrB_Index min_label = (GrB_Index)-1;
            for (GrB_Index j = 0; j < local_n; j++)
            {
                if (max_count < local_bin_count[j])
                {
                    max_count = local_bin_count[j];
                    min_label = local_bin_label[j];
                }
                else if (max_count == local_bin_count[j] && min_label > local_bin_label[j] && local_bin_label[j] != (GrB_Index)-1)
                {
                    min_label = local_bin_label[j];
                }
            }
            if (neighbor_n > LOCAL_BIN_SIZE)
            {
                for (GrB_Index j = bin_base; j < bin_base + neighbor_n - LOCAL_BIN_SIZE; j++)
                {
                    if (max_count < bin_count[j])
                    {
                        max_count = bin_count[j];
                        min_label = bin_label[j];
                    }
                    else if (max_count == bin_count[j] && min_label > bin_label[j] && bin_label[j] != (GrB_Index)-1)
                    {
                        min_label = bin_label[j];
                    }
                }
            }

            // 3. Update label
            if (min_label != (GrB_Index)-1)
            {
                // labels[srcNode] = min_label; // TODO: potential overflow
                new_labels[srcNode] = min_label;
            }
        }
        __syncthreads();
    }
}

__host__ void cdlp_gpu(GrB_Index *Ap, GrB_Index Ap_size, GrB_Index *Aj, GrB_Index Aj_size, GrB_Vector *CDLP_handle, GrB_Index N, bool symmetric, int itermax)
{
    GrB_Index *Ap_k;
    GrB_Index *Aj_k;
    GrB_Index *labels_k;
    GrB_Index *new_labels_k;
    GrB_Index *labels;
    GrB_Index *bin_count_k, *bin_label_k; // For dynamically counting labels (can be optimized using shared memory plus a overflow global memory)
    int is_equal = 1;
    int *is_equal_k;

    hipMalloc((void **)&Ap_k, Ap_size);
    hipMalloc((void **)&Aj_k, Aj_size);
    hipMalloc((void **)&labels_k, N * sizeof(GrB_Index));
    hipMalloc((void **)&new_labels_k, N * sizeof(GrB_Index));
    hipHostMalloc((void **)&labels, N * sizeof(GrB_Index));
    hipMalloc((void **)&bin_count_k, Aj_size);
    hipMalloc((void **)&bin_label_k, Aj_size);
    hipMalloc((void **)&is_equal_k, sizeof(int));

#if DEBUG_PRINT != 0
    // PRINT("FINISH CUDA MALLOC");
    std::cout<<"FINISH CUDA MALLOC"<<std::endl;
#endif

#if optimized1
    int *bin_index;
    hipMalloc((void **)&bin_index, sizeof(int));
    hipMemset(bin_index, 0, sizeof(int));
#endif

    hipMemcpy(Ap_k, Ap, Ap_size, hipMemcpyHostToDevice);
    hipMemcpy(Aj_k, Aj, Aj_size, hipMemcpyHostToDevice);

#if DEBUG_PRINT != 0
    // PRINT("FINISH CUDA MEMCPY");
    std::cout<<"FINISH CUDA MEMCPY"<<std::endl;
#endif

    dim3 DimGrid(GRID_DIM, 1, 1);
    dim3 DimBlock(BLOCK_DIM, 1);

    initialize_label<<<DimGrid, DimBlock>>>(labels_k, N);

    timer_start("CDLP_GPU MAIN LOOP USING CUDA KERNEL");
    for (int i = 0; i < itermax; ++i)
    {
        // PRINT("RUNNING ITERATION {}", i);
#if DEBUG_PRINT != 0
        std::cout<<"RUNNING ITERATION "<<i<<std::endl;
#endif

#if optimized1
        cdlp_optimized1<<<DimGrid, DimBlock>>>(Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, bin_count_k, bin_label_k, bin_index);
#else
        cdlp_base<<<DimGrid, DimBlock>>>(Ap_k, Aj_k, labels_k, new_labels_k, N, symmetric, bin_count_k, bin_label_k);
#endif
        // hipDeviceSynchronize();
        hipMemset(is_equal_k, 1, sizeof(int));
        check_equality<<<DimGrid, DimBlock>>>(labels_k, new_labels_k, N, is_equal_k);
        // hipDeviceSynchronize();
        hipMemcpy(&is_equal, is_equal_k, sizeof(int), hipMemcpyDeviceToHost);
        if (is_equal)
            break;
        else
        {
            // hipMemcpy(labels_k, new_labels_k, N * sizeof(GrB_Index), hipMemcpyDeviceToDevice);
            // optimization: double buffering, avoid memcpy
            GrB_Index *tmp = labels_k;
            labels_k = new_labels_k;
            new_labels_k = tmp;
        }
    }
    hipDeviceSynchronize();

    timer_stop();

    hipMemcpy(labels, labels_k, N * sizeof(GrB_Index), hipMemcpyDeviceToHost);

#if DEBUG_PRINT != 0
    // PRINT("RUNNING CUDA FREE");
    std::cout<<"RUNNING CUDA FREE"<<std::endl;
#endif

    hipFree(Ap_k);
    hipFree(Aj_k);
    hipFree(labels_k);
    hipFree(new_labels_k);
#if optimized1
    hipFree(bin_index);
#endif

#if DEBUG_PRINT != 0
    // PRINT("CONVERT TO GRB_VECTOR");
    std::cout<<"CONVERT TO GRB_VECTOR"<<std::endl;
#endif

    GrB_Vector CDLP = NULL;
    GrB_Vector_new(&CDLP, GrB_UINT64, N);
    for (GrB_Index i = 0; i < N; i++)
    {
        GrB_Vector_setElement_UINT64(CDLP, labels[i], i);
    }
    (*CDLP_handle) = CDLP;

    hipFree(labels);
}